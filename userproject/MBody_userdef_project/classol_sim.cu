/*--------------------------------------------------------------------------
   Author: Thomas Nowotny
  
   Institute: Institute for Nonlinear Science
              University of California San Diego
              La Jolla, CA 92093-0402
  
   email to:  tnowotny@ucsd.edu
  
   initial version: 2002-09-26
  
--------------------------------------------------------------------------*/

//--------------------------------------------------------------------------
/*! \file classol_sim.cu

\brief Main entry point for the classol (CLASSification in OLfaction) model simulation. Provided as a part of the complete example of simulating the MBody1 mushroom body model. 
*/
//--------------------------------------------------------------------------


#include "classol_sim.h"

//--------------------------------------------------------------------------
/*! \brief This function is the entry point for running the simulation of the MBody1 model network.
*/
//--------------------------------------------------------------------------


int main(int argc, char *argv[])
{
  if (argc != 3)
  {
    fprintf(stderr, "usage: classol_sim <basename> <CPU=0, GPU=1> \n");
    return 1;
  }
  int which= atoi(argv[2]);
  string OutDir = toString(argv[1]) +"_output";
  string name;
  name= OutDir+ "/"+ toString(argv[1]) + toString(".time");
  FILE *timef= fopen(name.c_str(),"a");  

  timer.startTimer();
  patSetTime= (int) (PAT_TIME/DT);
  patFireTime= (int) (PATFTIME/DT);
  fprintf(stdout, "# DT %f \n", DT);
  fprintf(stdout, "# T_REPORT_TME %f \n", T_REPORT_TME);
  fprintf(stdout, "# SYN_OUT_TME %f \n",  SYN_OUT_TME);
  fprintf(stdout, "# PATFTIME %f \n", PATFTIME); 
  fprintf(stdout, "# patFireTime %d \n", patFireTime);
  fprintf(stdout, "# PAT_TIME %f \n", PAT_TIME);
  fprintf(stdout, "# patSetTime %d \n", patSetTime);
  fprintf(stdout, "# TOTAL_TME %f \n", TOTAL_TME);
  
  name= OutDir+ "/"+ toString(argv[1]) + toString(".out.Vm"); 
  FILE *osf= fopen(name.c_str(),"w");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".out.St"); 
  FILE *osf2= fopen(name.c_str(),"w");
  


  //-----------------------------------------------------------------
  // build the neuronal circuitery
  classol locust;

  fprintf(stdout, "# reading PN-KC synapses ... \n");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".pnkc");
  FILE *f= fopen(name.c_str(),"r");
  locust.read_pnkcsyns(f);
  fclose(f);
 
  fprintf(stdout, "# reading PN-LHI synapses ... \n");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".pnlhi");
  f= fopen(name.c_str(), "r");
  locust.read_pnlhisyns(f);
  fclose(f);   
  
  fprintf(stdout, "# reading KC-DN synapses ... \n");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".kcdn");
  f= fopen(name.c_str(), "r");
  locust.read_kcdnsyns(f);

  fprintf(stdout, "# reading input patterns ... \n");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".inpat");
  f= fopen(name.c_str(), "r");
  locust.read_input_patterns(f);
  fclose(f);
  locust.generate_baserates();

  if (which == GPU) {
    locust.allocate_device_mem_patterns();
  }
  locust.init(which);         // this includes copying g's for the GPU version

  fprintf(stdout, "# neuronal circuitery built, start computation ... \n\n");

  //------------------------------------------------------------------
  // output general parameters to output file and start the simulation

  fprintf(stdout, "# We are running with fixed time step %f \n", DT);
  fprintf(stdout, "# initial wait time execution ... \n");

  t= 0.0;
  int done= 0;
  float last_t_report=  t;
//  locust.output_state(os, which);  
//  locust.output_spikes(os, which);  
  locust.run(DT, which);
//  locust.output_state(os, which);  
//  float synwriteT= 0.0f;
//  int synwrite= 0;
//  unsigned int sum= 0;
  while (!done) 
  {
    if (which == GPU) {
      locust.getSpikeNumbersFromGPU();
      locust.getSpikesFromGPU();
    }
//    if (which == GPU) locust.getSpikeNumbersFromGPU();
    locust.run(DT, which); // run next batch
    if (which == GPU) {  
     CHECK_CUDA_ERRORS(hipMemcpy(VDN, d_VDN, 10*sizeof(float), hipMemcpyDeviceToHost));
    }
    locust.sum_spikes();
//    locust.output_spikes(osf, which);
//    locust.output_state(os, which);  // while outputting the current one ...

   
    locust.output_spikes(osf2, which);

    fprintf(osf, "%f ", t);
    for (int i= 0; i < 10; i++) {
    fprintf(osf, "%f ", VDN[i]);
   }
    fprintf(osf,"\n");

    // report progress
    if (t - last_t_report >= T_REPORT_TME)
    {
      fprintf(stdout, "time %f \n", t);
      last_t_report= t;
    }
    // output synapses occasionally
    // if (synwrite) {
    //   lastsynwrite= synwriteT;
    //   name= toString(argv[1]) + toString(".") + toString((int) synwriteT);
    //   name+= toString(".syn");
    //   f= fopen(name.c_str(),"w");
    //   locust.write_kcdnsyns(f);
    //   fclose(f);
    //   synwrite= 0;
    // }
    // if (t - lastsynwrite >= SYN_OUT_TME) {
    //   locust.get_kcdnsyns();
    //   synwrite= 1;
    // ¯  synwriteT= t;
    // }
    done= (t >= TOTAL_TME);
  }
//  locust.output_state(os);
//    if (which == GPU) locust.getSpikesFromGPU();
//    locust.output_spikes(os, which);
  // if (synwrite) {
  //   lastsynwrite= t;
  //   name= toString(argv[1]) + toString(".") + toString((int) t);
  //   name+= toString(".syn");
  //   f= fopen(name.c_str());
  //   locust.write_kcdnsyns(f);
  // fclose(f);
  //   synwrite= 0;
  // }

  timer.stopTimer();
  cerr << "output files are created under the current directory." << endl;
  fprintf(timef, "%d %u %u %u %u %u %.4f %.2f %.1f %.2f\n",which, locust.model.sumNeuronN[locust.model.neuronGrpN-1], locust.sumPN, locust.sumKC, locust.sumLHI, locust.sumDN, timer.getElapsedTime(),VDN[0], TOTAL_TME, DT);
  fclose(osf);
  fclose(osf2);
  fclose(timef);
	freeDeviceMem();
  hipDeviceReset();
  return 0;
}
