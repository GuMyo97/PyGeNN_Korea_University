/*--------------------------------------------------------------------------
   Author: Thomas Nowotny
  
   Institute: Institute for Nonlinear Dynamics
              University of California San Diego
              La Jolla, CA 92093-0402
  
   email to:  tnowotny@ucsd.edu
  
   initial version: 2002-09-26
  
--------------------------------------------------------------------------*/

#include "classol_sim.h"

int main(int argc, char *argv[])
{
  if (argc != 3)
  {
    fprintf(stderr, "usage: classol_sim <basename> <CPU=0, GPU=1> \n");
    return 1;
  }
  int which= atoi(argv[2]);
  string name;
  name= toString(argv[1]) + toString(".time");
  FILE *timef= fopen(name.c_str(),"w");  

  timer.startTimer();
  

  fprintf(stderr, "# DT %f \n", DT);
  fprintf(stderr, "# T_REPORT_TME %f \n", T_REPORT_TME);
  fprintf(stderr, "# SYN_OUT_TME %f \n",  SYN_OUT_TME);
  fprintf(stderr, "# PATFTIME %f \n", PATFTIME); 
  fprintf(stderr, "# PAT_FIRETIME %d \n", PAT_FIRETIME);
  fprintf(stderr, "# PAT_TIME %f \n", PAT_TIME);
  fprintf(stderr, "# PAT_SETTIME %d \n", PAT_SETTIME);
  fprintf(stderr, "# TOTAL_TME %d \n", TOTAL_TME);
  
  name= toString(argv[1]) + toString(".out.st"); 
  FILE *osf= fopen(name.c_str(),"w");

  //-----------------------------------------------------------------
  // build the neuronal circuitery
  classol locust;

  fprintf(stderr, "# reading PN-KC synapses ... \n");
  name= toString(argv[1]) + toString(".pnkc");
  FILE *f= fopen(name.c_str(),"r");
  locust.read_pnkcsyns(f);
  fclose(f);   
 
  fprintf(stderr, "# reading PN-LHI synapses ... \n");
  name= toString(argv[1]) + toString(".pnlhi");
  f= fopen(name.c_str(), "r");
  locust.read_pnlhisyns(f);
  fclose(f);   
  
  fprintf(stderr, "# reading KC-DN synapses ... \n");
  name= toString(argv[1]) + toString(".kcdn");
  f= fopen(name.c_str(), "r");
  locust.read_kcdnsyns(f);
   
  fprintf(stderr, "# reading input patterns ... \n");
  name= toString(argv[1]) + toString(".inpat");
  f= fopen(name.c_str(), "r");
  locust.read_input_patterns(f);
  fclose(f);
  locust.generate_baserates();

  if (which == GPU) {
    locust.allocate_device_mem_patterns();
  }
  locust.init(which);         // this includes copying g's for the GPU version

  fprintf(stderr, "# neuronal circuitery built, start computation ... \n\n");

  //------------------------------------------------------------------
  // output general parameters to output file and start the simulation

  fprintf(stderr, "# We are running with fixed time step %f \n", DT);
  fprintf(stderr, "# initial wait time execution ... \n");

  t= 0.0;
  void *devPtr;
//  float lastsynwrite= t;
  int done= 0;
  float last_t_report=  t;
//  locust.output_state(os, which);  
//  locust.output_spikes(os, which);  
  locust.run(DT, which);
//locust.output_state(os, which);  
//  float synwriteT= 0.0f;
//  int synwrite= 0;
unsigned int sum= 0;
  while (!done) 
  {
//   if (which == GPU) locust.getSpikesFromGPU();
//    if (which == GPU) locust.getSpikeNumbersFromGPU();
    locust.run(DT, which); // run next batch
    if (which == GPU) {  
      hipGetSymbolAddress(&devPtr, HIP_SYMBOL("d_VDN"));
      CUDA_SAFE_CALL(hipMemcpy(VDN, devPtr, 10*sizeof(float), hipMemcpyDeviceToHost));
    }
//    locust.sum_spikes();
//    locust.output_spikes(os, which);
//   locust.output_state(os, which);  // while outputting the current one ...
   fprintf(osf, "%f ", t);
   for (int i= 0; i < 10; i++) {
     fprintf(osf, "%f ", VDN[i]);
   }
   fprintf(osf,"\n");
//      hipDeviceSynchronize();

   // report progress
    if (t - last_t_report >= T_REPORT_TME)
    {
      fprintf(stderr, "time %f \n", t);
      last_t_report= t;
      //locust.output_state(os);
    }
    // output synapses occasionally
    // if (synwrite) {
    //   lastsynwrite= synwriteT;
    //   name= toString(argv[1]) + toString(".") + toString((int) synwriteT);
    //   name+= toString(".syn");
    //   f= fopen(name.c_str(),"w");
    //   locust.write_kcdnsyns(f);
    //   fclose(f);
    //   synwrite= 0;
    // }
    // if (t - lastsynwrite >= SYN_OUT_TME) {
    //   locust.get_kcdnsyns();
    //   synwrite= 1;
    //   synwriteT= t;
    // }
    done= (t >= TOTAL_TME);
  }
//  locust.output_state(os);
//    if (which == GPU) locust.getSpikesFromGPU();
//    locust.output_spikes(os, which);
  // if (synwrite) {
  //   lastsynwrite= t;
  //   name= toString(argv[1]) + toString(".") + toString((int) t);
  //   name+= toString(".syn");
  //   f= fopen(name.c_str());
  //   locust.write_kcdnsyns(f);
  // fclose(f);
  //   synwrite= 0;
  // }

  timer.stopTimer();
  fprintf(timef, "%d %d %d %d %f \n", locust.sumPN, locust.sumKC, locust.sumLHI, locust.sumDN, timer.getElapsedTime());

  return 0;
}
